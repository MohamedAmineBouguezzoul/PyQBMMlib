#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cudaErr.hpp"
#include "main.hpp"

/**********************************
 * gpu kernels 
 */

__global__ void c20_kernel(float* M, float* c20, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c20[idx] = ((M[6*idx+3] * M[6*idx]) - (M[6*idx+1] * M[6*idx+1])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c11_kernel(float* M, float* c11, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c11[idx] = ((M[6*idx+4] * M[6*idx]) - (M[6*idx+1] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c02_kernel(float* M, float* c02, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c02[idx] = ((M[6*idx+5] * M[6*idx]) - (M[6*idx+2] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void init_M(float* value, float* M, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        M[3*idx] = 1;
        M[3*idx+1] = 0;
        M[3*idx+2] = value[idx];
        idx += blockDim.x;
    };
};

__global__ void nu_kernel(float* c11, float* c20, float* xi, float* nu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float c = c11[idx]/c20[idx];
        nu[2*idx] = c*xi[2*idx];
        nu[2*idx+1] = c*xi[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void mu_kernel(float* c02, float* nu, float* w, float* mu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        mu[idx] = c02[idx] - (w[2*idx]*nu[2*idx]*nu[2*idx] 
                    + w[2*idx+1]*nu[2*idx+1]*nu[2*idx+1]);
        idx += blockDim.x;
    };
};

__global__ void mu_kernel2(float* c02, float* c11, float* c20, float* mu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        mu[idx] = c02[idx] - c11[idx]*c11[idx]/c20[idx];
        idx += blockDim.x;
    };
};

__global__ void hyqmom2_kernel(float* M, float* w, float* x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float C2 = ((M[3*idx] * M[3*idx+2]) - (M[3*idx+1] * M[3*idx+1])) 
                    / (M[3*idx] * M[3*idx]);
        w[2*idx] = M[3*idx]/2;
        w[2*idx+1] = M[3*idx]/2;
        x[2*idx] = (M[3*idx+1]/M[3*idx]) - sqrt(C2);
        x[2*idx+1] = (M[3*idx+1]/M[3*idx]) + sqrt(C2);
        idx += blockDim.x;
    };
};

__global__ void weight_kernel(float* M, float* w1, float* w2, float* w_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        w_final[4*idx] = M[6*idx] * w1[2*idx] * w2[2*idx];
        w_final[4*idx+1] = M[6*idx] * w1[2*idx] * w2[2*idx+1];
        w_final[4*idx+2] = M[6*idx] * w1[2*idx+1] * w2[2*idx];
        w_final[4*idx+3] = M[6*idx] * w1[2*idx+1] * w2[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void x_kernel(float* M, float* x1, float* x_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        x_final[4*idx] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+1] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+2] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        x_final[4*idx+3] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void y_kernel(float* M, float* nu, float* x2, float* y_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        y_final[4*idx] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx];
        y_final[4*idx+1] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx+1];
        y_final[4*idx+2] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx];
        y_final[4*idx+3] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx+1];
        idx += blockDim.x;
    };
};

float qmom_cuda(float moments[], int num_moments, float* result) {

    // timer for measuring kernel execution time
    // measurement done in miliseconds
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("[CUDA] Setting up memmory... \n");
    float *moments_gpu;
    float *c20, *c11, *c02; 
    float *M_inter, *w_inter_1, *w_inter_2, *x_inter_1, *x_inter_2;
    float *nu, *mu;
    float *w_final_gpu, *x_final_gpu, *y_final_gpu;
    float *w_final_cpu, *x_final_cpu, *y_final_cpu;
    
    //// allocate host memory
    w_final_cpu = new float[num_moments*4];
    x_final_cpu = new float[num_moments*4];
    y_final_cpu = new float[num_moments*4];

    //// allocate device memory 
    // input
    gpuErrchk(hipMalloc(&moments_gpu, sizeof(float)*num_moments*6));
    // Central moments
    gpuErrchk(hipMalloc(&c20, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c11, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c02, sizeof(float)*num_moments));
    // intermediate M, w, x as input and output of HyQMOM2
    gpuErrchk(hipMalloc(&M_inter, sizeof(float)*num_moments*3));
    gpuErrchk(hipMalloc(&w_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&w_inter_2, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_2, sizeof(float)*num_moments*2));
    // intermediate values 
    gpuErrchk(hipMalloc(&nu, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&mu, sizeof(float)*num_moments));
    // final weight, abscissas: 
    gpuErrchk(hipMalloc(&w_final_gpu, sizeof(float)*num_moments*4));
    gpuErrchk(hipMalloc(&x_final_gpu, sizeof(float)*num_moments*4));
    hipMalloc(&y_final_gpu, sizeof(float)*num_moments*4);  
    //copy input from host to device 
    gpuErrchk(hipMemcpy(moments_gpu, moments, 
                    sizeof(float)*num_moments*6, hipMemcpyHostToDevice)
    );


    // thread block is set to be 1D,
    int num_threads = 1024;
    int num_blocks = 1;

    // set up three streams for concurrent kernels
    hipStream_t stream1, stream2, stream3;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking);

    //// Calculating 
    printf("[CUDA] starting calculation. Timer on ... \n");
    hipEventRecord(start); //start the timer

    // Central moments set_M_kernel
    c11_kernel<<<num_blocks, num_threads, 0, stream1>>>(moments_gpu, c11, num_moments);
    c20_kernel<<<num_blocks, num_threads, 0, stream2>>>(moments_gpu, c20, num_moments);
    c02_kernel<<<num_blocks, num_threads, 0, stream3>>>(moments_gpu, c02, num_moments);
    init_M<<<num_blocks, num_threads, 0, stream3>>>(c02, M_inter, num_moments);

    hyqmom2_kernel<<<num_blocks, num_threads, 0, stream3>>>(M_inter, w_inter_1, x_inter_1, num_moments);
    nu_kernel<<<num_blocks, num_threads, 0, stream3>>>(c11, c20, x_inter_1, nu, num_moments);
    mu_kernel2<<<num_blocks, num_threads, 0, stream2>>>(c02, c11, c20, mu, num_moments);
    init_M<<<num_blocks, num_threads, 0, stream2>>>(mu, M_inter, num_moments);

    // second hyqmom2
    hyqmom2_kernel<<<num_blocks, num_threads, 0, stream2>>>(M_inter, w_inter_2, x_inter_2, num_moments);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    // final results
    weight_kernel<<<num_blocks, num_threads, 0, stream1>>>(moments_gpu, w_inter_1, w_inter_2, w_final_gpu, num_moments);
    x_kernel<<<num_blocks, num_threads, 0, stream2>>>(moments_gpu, x_inter_1, x_final_gpu, num_moments);
    y_kernel<<<num_blocks, num_threads, 0, stream3>>>(moments_gpu, nu, x_inter_2, y_final_gpu, num_moments);
    hipEventRecord(stop); //stop the timer
    printf("[CUDA] Finished calculation. Timer off... \n");

    // copy result from device to host 
    gpuErrchk(hipMemcpyAsync(result, w_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost, stream1));
    gpuErrchk(hipMemcpyAsync(x_final_cpu, x_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost, stream2));
    gpuErrchk(hipMemcpyAsync(y_final_cpu, y_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost, stream3));
    // --TODO-- verify the result somehow? 

    float calc_duration; 
    hipEventElapsedTime(&calc_duration, start, stop);

    hipFree(moments_gpu);
    hipFree(c20);
    hipFree(c02);
    hipFree(c11);
    hipFree(M_inter);
    hipFree(w_inter_1);
    hipFree(w_inter_2);
    hipFree(x_inter_1);
    hipFree(x_inter_2);
    hipFree(mu);
    hipFree(nu);
    hipFree(w_final_gpu);
    hipFree(x_final_gpu);
    hipFree(y_final_gpu);
    delete[] w_final_cpu, x_final_cpu, y_final_cpu;

    return calc_duration;
}