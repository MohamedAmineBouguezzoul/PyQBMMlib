#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cudaErr.hpp"
#include "main.hpp"

/**********************************
 * gpu kernels 
 */

__global__ void c20_kernel(float* M, float* c20, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c20[idx] = ((M[6*idx+3] * M[6*idx]) - (M[6*idx+1] * M[6*idx+1])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c11_kernel(float* M, float* c11, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c11[idx] = ((M[6*idx+4] * M[6*idx]) - (M[6*idx+1] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c02_kernel(float* M, float* c02, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c02[idx] = ((M[6*idx+5] * M[6*idx]) - (M[6*idx+2] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void init_M(float* value, float* M, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        M[3*idx] = 1;
        M[3*idx+1] = 0;
        M[3*idx+2] = value[idx];
        idx += blockDim.x;
    };
};

__global__ void nu_kernel(float* c11, float* c20, float* xi, float* nu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float c = c11[idx]/c20[idx];
        nu[2*idx] = c*xi[2*idx];
        nu[2*idx+1] = c*xi[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void mu_kernel(float* c02, float* nu, float* w, float* mu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        mu[idx] = c02[idx] - (w[2*idx]*nu[2*idx]*nu[2*idx] 
                    + w[2*idx+1]*nu[2*idx+1]*nu[2*idx+1]);
        idx += blockDim.x;
    };
};

__global__ void hyqmom2_kernel(float* M, float* w, float* x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float C2 = ((M[3*idx] * M[3*idx+2]) - (M[3*idx+1] * M[3*idx+1])) 
                    / (M[3*idx] * M[3*idx]);
        w[2*idx] = M[3*idx]/2;
        w[2*idx+1] = M[3*idx]/2;
        x[2*idx] = (M[3*idx+1]/M[3*idx]) - sqrt(C2);
        x[2*idx+1] = (M[3*idx+1]/M[3*idx]) + sqrt(C2);
        idx += blockDim.x;
    };
};

__global__ void weight_kernel(float* M, float* w1, float* w2, float* w_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        w_final[4*idx] = M[6*idx] * w1[2*idx] * w2[2*idx];
        w_final[4*idx+1] = M[6*idx] * w1[2*idx] * w2[2*idx+1];
        w_final[4*idx+2] = M[6*idx] * w1[2*idx+1] * w2[2*idx];
        w_final[4*idx+3] = M[6*idx] * w1[2*idx+1] * w2[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void x_kernel(float* M, float* x1, float* x_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        x_final[4*idx] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+1] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+2] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        x_final[4*idx+3] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void y_kernel(float* M, float* nu, float* x2, float* y_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        y_final[4*idx] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx];
        y_final[4*idx+1] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx+1];
        y_final[4*idx+2] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx];
        y_final[4*idx+3] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx+1];
        idx += blockDim.x;
    };
};

float qmom_cuda(float moments[], int num_moments,
    float xout[], float yout[], float wout[]) {

    // timer for measuring kernel execution time
    // measurement done in miliseconds
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //// Setup memory 
    printf("[CUDA] Setting up memmory... \n");
    float *moments_gpu;
    float *c20, *c11, *c02; 
    float *M_inter, *w_inter_1, *w_inter_2, *x_inter_1, *x_inter_2;
    float *nu, *mu;
    float *w_final_gpu, *x_final_gpu, *y_final_gpu;    

    //// allocate device memory 
    // input
    gpuErrchk(hipMalloc(&moments_gpu, sizeof(float)*num_moments*6));
    // Central moments
    gpuErrchk(hipMalloc(&c20, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c11, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c02, sizeof(float)*num_moments));
    // intermediate M, w, x as input and output of HyQMOM2
    gpuErrchk(hipMalloc(&M_inter, sizeof(float)*num_moments*3));
    gpuErrchk(hipMalloc(&w_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&w_inter_2, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_2, sizeof(float)*num_moments*2));
    // intermediate values 
    gpuErrchk(hipMalloc(&nu, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&mu, sizeof(float)*num_moments));
    // final weight, abscissas: 
    gpuErrchk(hipMalloc(&w_final_gpu, sizeof(float)*num_moments*4));
    gpuErrchk(hipMalloc(&x_final_gpu, sizeof(float)*num_moments*4));
    hipMalloc(&y_final_gpu, sizeof(float)*num_moments*4);  
    //copy input from host to device 
    gpuErrchk(hipMemcpy(moments_gpu, moments, 
                    sizeof(float)*num_moments*6, hipMemcpyHostToDevice)
    );


    // thread block is set to be 1D,
    int num_threads = 1024;

    //// Calculating 
    printf("[CUDA] starting calculation. Timer on ... \n");
    hipEventRecord(start); //start the timer

    // Central moments set_M_kernel
    c11_kernel<<<1, num_threads>>>(moments_gpu, c11, num_moments);
    c20_kernel<<<1, num_threads>>>(moments_gpu, c20, num_moments);
    c02_kernel<<<1, num_threads>>>(moments_gpu, c02, num_moments);

    // first hyqmom2
    init_M<<<1, num_threads>>>(c02, M_inter, num_moments);
    hyqmom2_kernel<<<1, num_threads>>>(M_inter, w_inter_1, x_inter_1, num_moments);
    
    // intermediate values 
    nu_kernel<<<1, num_threads>>>(c11, c20, x_inter_1, nu, num_moments);
    mu_kernel<<<1, num_threads>>>(c02, nu, w_inter_1, mu, num_moments);
    __global__ void x_kernel(float* M, float* x1, float* x_final, int N) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x*2;
    
        while (idx < N) {
    
            float4 *x_final4 = reinterpret_cast<float4*>(&(x_final[idx*4]));
            float2 *M6_2 = reinterpret_cast<float2*>(&(M[idx*6]));
            float4 *x1_4 = reinterpret_cast<float4*>(&(x1[idx*2]));
    
            float4 temp_final1;
            float4 temp_final2;
            float2 temp_M6 = M6_2[0];
            float2 temp_M12 = M6_2[3];
            float4 temp_x1 = x1_4[0];
    
            float quotient1 = temp_M6.y/temp_M6.x;
            float quotient2 = temp_M12.y/temp_M12.x;
    
            temp_final1.x = quotient1 + temp_x1.x;
            temp_final1.y = temp_final1.x;
            temp_final1.z = quotient1 + temp_x1.y;
            temp_final1.w = temp_final1.z;
            temp_final2.x = quotient2 + temp_x1.z;
            temp_final2.y = temp_final2.x;
            temp_final2.z = quotient2 + temp_x1.w;
            temp_final2.w = temp_final2.z;
    
            x_final4[0] = temp_final1;
            x_final4[1] = temp_final2;
            
            // printf("[thread %d] temp_x1: %f %f %f %f \n" , idx, temp_x1.x, temp_x1.y, temp_x1.z, temp_x1.w);
            // printf("[thread %d] quotient: %f \n",idx, quotient2);
            // printf("[thread %d] temp_final2: %f %f %f %f \n" , idx, temp_final2.x, temp_final2.y, temp_final2.z, temp_final2.w);
            // x_final[4*idx] = M[6*idx+1]/M[6*idx] + x1[2*idx];
            // x_final[4*idx+1] = M[6*idx+1]/M[6*idx] + x1[2*idx];
            // x_final[4*idx+2] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
            // x_final[4*idx+3] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
            idx += 2*blockDim.x;
        };
    };eads>>>(M_inter, w_inter_2, x_inter_2, num_moments);

    // final results
    weight_kernel<<<1, num_threads>>>(moments_gpu, w_inter_1, w_inter_2, w_final_gpu, num_moments);
    x_kernel<<<1, num_threads>>>(moments_gpu, x_inter_1, x_final_gpu, num_moments);
    y_kernel<<<1, num_threads>>>(moments_gpu, nu, x_inter_2, y_final_gpu, num_moments);
    hipEventRecord(stop); //stop the timer
    printf("[CUDA] Finished calculation. Timer off... \n");

    // copy result from device to host 
    gpuErrchk(hipMemcpy(wout, w_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(xout, x_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(yout, y_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    // --TODO-- verify the result somehow? 

    float calc_duration; 
    hipEventElapsedTime(&calc_duration, start, stop);

    hipFree(moments_gpu);
    hipFree(c20);
    hipFree(c02);
    hipFree(c11);
    hipFree(M_inter);
    hipFree(w_inter_1);
    hipFree(w_inter_2);
    hipFree(x_inter_1);
    hipFree(x_inter_2);
    hipFree(mu);
    hipFree(nu);
    hipFree(w_final_gpu);
    hipFree(x_final_gpu);
    hipFree(y_final_gpu);

    return calc_duration;
}