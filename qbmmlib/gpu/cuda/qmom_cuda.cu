#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cudaErr.hpp"
#include "main.hpp"

/**********************************
 * gpu kernels 
 */

__global__ void c20_kernel(float* M, float* c20, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c20[idx] = ((M[6*idx+3] * M[6*idx]) - (M[6*idx+1] * M[6*idx+1])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c11_kernel(float* M, float* c11, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c11[idx] = ((M[6*idx+4] * M[6*idx]) - (M[6*idx+1] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void c02_kernel(float* M, float* c02, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        c02[idx] = ((M[6*idx+5] * M[6*idx]) - (M[6*idx+2] * M[6*idx+2])) 
                    / (M[6*idx] * M[6*idx]);
        idx += blockDim.x;
    };
};

__global__ void init_M(float* value, float* M, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        M[3*idx] = 1;
        M[3*idx+1] = 0;
        M[3*idx+2] = value[idx];
        idx += blockDim.x;
    };
};

__global__ void nu_kernel(float* c11, float* c20, float* xi, float* nu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float c = c11[idx]/c20[idx];
        nu[2*idx] = c*xi[2*idx];
        nu[2*idx+1] = c*xi[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void mu_kernel(float* c02, float* nu, float* w, float* mu, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        mu[idx] = c02[idx] - (w[2*idx]*nu[2*idx]*nu[2*idx] 
                    + w[2*idx+1]*nu[2*idx+1]*nu[2*idx+1]);
        idx += blockDim.x;
    };
};

__global__ void hyqmom2_kernel(float* M, float* w, float* x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        float C2 = ((M[3*idx] * M[3*idx+2]) - (M[3*idx+1] * M[3*idx+1])) 
                    / (M[3*idx] * M[3*idx]);
        w[2*idx] = M[3*idx]/2;
        w[2*idx+1] = M[3*idx]/2;
        x[2*idx] = (M[3*idx+1]/M[3*idx]) - sqrt(C2);
        x[2*idx+1] = (M[3*idx+1]/M[3*idx]) + sqrt(C2);
        idx += blockDim.x;
    };
};

__global__ void weight_kernel(float* M, float* w1, float* w2, float* w_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        w_final[4*idx] = M[6*idx] * w1[2*idx] * w2[2*idx];
        w_final[4*idx+1] = M[6*idx] * w1[2*idx] * w2[2*idx+1];
        w_final[4*idx+2] = M[6*idx] * w1[2*idx+1] * w2[2*idx];
        w_final[4*idx+3] = M[6*idx] * w1[2*idx+1] * w2[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void x_kernel(float* M, float* x1, float* x_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        x_final[4*idx] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+1] = M[6*idx+1]/M[6*idx] + x1[2*idx];
        x_final[4*idx+2] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        x_final[4*idx+3] = M[6*idx+1]/M[6*idx] + x1[2*idx+1];
        idx += blockDim.x;
    };
};

__global__ void y_kernel(float* M, float* nu, float* x2, float* y_final, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        y_final[4*idx] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx];
        y_final[4*idx+1] = M[6*idx+2]/M[6*idx] + nu[2*idx] + x2[2*idx+1];
        y_final[4*idx+2] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx];
        y_final[4*idx+3] = M[6*idx+2]/M[6*idx] + nu[2*idx+1] + x2[2*idx+1];
        idx += blockDim.x;
    };
};

float qmom_cuda(float moments[], int num_moments,
    float xout[], float yout[], float wout[]) {

    // timer for measuring kernel execution time
    // measurement done in miliseconds
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //// Setup memory 
    printf("[CUDA] Setting up memmory... \n");
    float *moments_gpu;
    float *c20, *c11, *c02; 
    float *M_inter, *w_inter_1, *w_inter_2, *x_inter_1, *x_inter_2;
    float *nu, *mu;
    float *w_final_gpu, *x_final_gpu, *y_final_gpu;    

    //// allocate device memory 
    // input
    gpuErrchk(hipMalloc(&moments_gpu, sizeof(float)*num_moments*6));
    // Central moments
    gpuErrchk(hipMalloc(&c20, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c11, sizeof(float)*num_moments));
    gpuErrchk(hipMalloc(&c02, sizeof(float)*num_moments));
    // intermediate M, w, x as input and output of HyQMOM2
    gpuErrchk(hipMalloc(&M_inter, sizeof(float)*num_moments*3));
    gpuErrchk(hipMalloc(&w_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&w_inter_2, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_1, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&x_inter_2, sizeof(float)*num_moments*2));
    // intermediate values 
    gpuErrchk(hipMalloc(&nu, sizeof(float)*num_moments*2));
    gpuErrchk(hipMalloc(&mu, sizeof(float)*num_moments));
    // final weight, abscissas: 
    gpuErrchk(hipMalloc(&w_final_gpu, sizeof(float)*num_moments*4));
    gpuErrchk(hipMalloc(&x_final_gpu, sizeof(float)*num_moments*4));
    hipMalloc(&y_final_gpu, sizeof(float)*num_moments*4);  
    //copy input from host to device 
    gpuErrchk(hipMemcpy(moments_gpu, moments, 
                    sizeof(float)*num_moments*6, hipMemcpyHostToDevice)
    );


    // thread block is set to be 1D,
    int num_threads = 1024;

    //// Calculating 
    printf("[CUDA] starting calculation. Timer on ... \n");
    hipEventRecord(start); //start the timer

    // Central moments set_M_kernel
    c11_kernel<<<1, num_threads>>>(moments_gpu, c11, num_moments);
    c20_kernel<<<1, num_threads>>>(moments_gpu, c20, num_moments);
    c02_kernel<<<1, num_threads>>>(moments_gpu, c02, num_moments);

    // first hyqmom2
    init_M<<<1, num_threads>>>(c02, M_inter, num_moments);
    hyqmom2_kernel<<<1, num_threads>>>(M_inter, w_inter_1, x_inter_1, num_moments);
    
    // intermediate values 
    nu_kernel<<<1, num_threads>>>(c11, c20, x_inter_1, nu, num_moments);
    mu_kernel<<<1, num_threads>>>(c02, nu, w_inter_1, mu, num_moments);

    // second hyqmom2
    init_M<<<1, num_threads>>>(mu, M_inter, num_moments);
    hyqmom2_kernel<<<1, num_threads>>>(M_inter, w_inter_2, x_inter_2, num_moments);

    // final results
    weight_kernel<<<1, num_threads>>>(moments_gpu, w_inter_1, w_inter_2, w_final_gpu, num_moments);
    x_kernel<<<1, num_threads>>>(moments_gpu, x_inter_1, x_final_gpu, num_moments);
    y_kernel<<<1, num_threads>>>(moments_gpu, nu, x_inter_2, y_final_gpu, num_moments);
    hipEventRecord(stop); //stop the timer
    printf("[CUDA] Finished calculation. Timer off... \n");

    // copy result from device to host 
    gpuErrchk(hipMemcpy(wout, w_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(xout, x_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(yout, y_final_gpu, sizeof(float)*num_moments*4, hipMemcpyDeviceToHost));
    // --TODO-- verify the result somehow? 

    float calc_duration; 
    hipEventElapsedTime(&calc_duration, start, stop);

    hipFree(moments_gpu);
    hipFree(c20);
    hipFree(c02);
    hipFree(c11);
    hipFree(M_inter);
    hipFree(w_inter_1);
    hipFree(w_inter_2);
    hipFree(x_inter_1);
    hipFree(x_inter_2);
    hipFree(mu);
    hipFree(nu);
    hipFree(w_final_gpu);
    hipFree(x_final_gpu);
    hipFree(y_final_gpu);

    return calc_duration;
}