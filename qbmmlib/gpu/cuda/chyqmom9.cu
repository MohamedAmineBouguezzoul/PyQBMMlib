#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <cstdio>
#include <cassert>
#include <chrono>

#include "hyqmom.hpp"

// a helper function for calculating nth moment 
__device__ float sum_pow(float rho[], float yf[], float n, const int len) {
    float sum = 0;
    for (int i = 0; i < len; i++) {
        sum += rho[i] * pow(yf[i], n); 
    }
    return sum;
}

// set a segment of memory to a specific value
static __global__ void float_value_set(float *addr, float value, int size) {
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        addr[idx] = value;
    }
}

static __global__ void chyqmom9_cmoments(
    const float moments[], 
    float c_moments[],
    const int size, 
    const int stride)
{
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        // copy moments to local registers
        float mom[10], cmom[7];
        mom[0] = moments[idx];
        // printf("[tIdx %d] mom[0] = %f\n", idx, mom[0]);
        // normalize mom by mom[0];
        // mom[i] = mom[i]/mom[0] for i !=0
        for (int n=1; n<10; n++) {
            mom[n] = moments[n * stride + idx] / mom[0];
            // printf("[tIdx %d] mom[%d] = %f\n", idx, n, mom[n]);
        }
        //compute central moments
        cmom[0] = mom[3] - mom[1] * mom[1];
        cmom[1] = mom[4] - mom[1] * mom[2];
        cmom[2] = mom[5] - mom[2] * mom[2];
        cmom[3] = mom[6] - 3*mom[1]*mom[3] + 2*mom[1]*mom[1]*mom[1];
        cmom[4] = mom[7] - 3*mom[2]*mom[5] + 2*mom[2]*mom[2]*mom[2];
        cmom[5] = mom[8] - 4*mom[1]*mom[6] + 6*mom[1]*mom[1]*mom[3] -
        3*mom[1]*mom[1]*mom[1]*mom[1];
        cmom[6] = mom[9] - 4*mom[2]*mom[7] + 6*mom[2]*mom[2]*mom[5] -
        3*mom[2]*mom[2]*mom[2]*mom[2];

        c_moments[idx] = cmom[0];
        c_moments[1*stride + idx] =cmom[1];
        c_moments[2*stride + idx] =cmom[2];
        c_moments[3*stride + idx] =cmom[3];
        c_moments[4*stride + idx] =cmom[4];
        c_moments[5*stride + idx] =cmom[5];
        c_moments[6*stride + idx] =cmom[6];
    }
}

static __global__ void chyqmom9_mu_yf(
    const float c_moments[], 
    const float xp[], 
    const float rho[],
    float yf[], 
    float mu[], 
    const int size, 
    const int stride) 
{
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        float c_local[5] = {
            c_moments[idx],             // c02
            c_moments[1*stride + idx],  // c11
            c_moments[2*stride + idx],  // c20
            c_moments[4*stride + idx],  // c03
            c_moments[6*stride + idx]   // c04
        };
        float mu_avg = c_local[2] - c_local[1]*c_local[1]/c_local[0];
        float rho_local[3] = {
            rho[idx],          
            rho[1*stride + idx], 
            rho[2*stride + idx]  
        };
        float yf_local[3] = {
            c_local[1] * xp[idx] / c_local[2],
            c_local[1] * xp[stride + idx] / c_local[2],
            c_local[1] * xp[2*stride + idx] / c_local[2]
        };
        yf[idx] = yf_local[0];
        yf[stride + idx] = yf_local[1];
        yf[2*stride + idx] = yf_local[2];

        // if mu > csmall
        float q = (c_local[3] - sum_pow(rho_local, yf_local, 3.0, 3)) / 
                    pow(mu_avg, (3.0 / 2.0));
        float eta = (c_local[4] - sum_pow(rho_local, yf_local, 4.0, 3) - 
                    6 * sum_pow(rho_local, yf_local, 2.0, 3) * mu_avg) / 
                    pow(mu_avg, 2.0);

        float mu3 = q * pow(mu_avg, 3/2);
        float mu4 = eta * mu_avg * mu_avg;

        mu[idx] = mu_avg;
        mu[stride + idx] = mu3;

        mu[2*stride + idx] = mu4;
    }
}

static __global__ void chyqmom9_wout(
    float moments[], 
    float rho_1[], 
    float rho_2[], 
    float w[],
    const int size,
    const int stride)
{
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        float r1[3], r2[3];
        float mom = moments[idx];
        for (int n=0; n<3; n++) {
            r1[n] = rho_1[n * stride + idx];
            r2[n] = rho_2[n * stride + idx];
        }
        
        for (int row = 0; row < 3; row ++) {
            for (int col = 0; col < 3; col ++) {
                w[(3*row + col) * stride + idx] = r1[row] * r2[col] * mom;
                // printf("[tIdx %d] w[%d] = %f \n", tIdx, (3*row + col) * stride + idx, w[(3*row + col) * stride + idx]);
            }
        }
    }
}

static __global__ void chyqmom9_xout(
    float moments[], 
    float xp[],
    float x[],
    const int size, 
    const int stride)
{
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        float x_local[3];
        float bx = moments[stride + idx] / moments[idx];
        for (int n = 0; n < 3; n++) {
            x_local[n] = xp[n * stride + idx];
        }
        for (int row = 0; row < 3; row ++) {
            float val = x_local[row] + bx;
            for (int col = 0; col < 3; col ++) {
                x[(3*row + col) * stride + idx] = val;
            }
        }
    }
}

static __global__ void chyqmom9_yout(
    float moments[], 
    float xp3[],
    float yf[],
    float y[],
    const int size,
    const int stride)
{
    const int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = tIdx; idx < size; idx+=blockDim.x*gridDim.x) {
        float x_local[3];
        float yf_local[3];
        
        for (int n = 0; n < 3; n++) {
            x_local[n] = xp3[n * stride + idx];
            yf_local[n]= yf[n * stride + idx];
        }
        float by = moments[2*stride + idx] / moments[idx];

        for (int row = 0; row < 3; row ++) {
            for (int col = 0; col < 3; col ++) {
                y[(3*row + col) * stride + idx] = yf_local[row] + x_local[col] + by;
            }
        }
    }
}

float chyqmom9(float moments[], const int size, float w[], float x[], float y[], const int batch_size) {

    using namespace std::chrono;

    float *moments_d, *w_out_d, *x_out_d, *y_out_d;
    float *c_moments, *mu, *yf;
    float *m1, *x1, *w1, *x2, *w2;

    // memory allocation
    gpuErrchk(hipMalloc(&moments_d, sizeof(float)*size*10));
    gpuErrchk(hipMalloc(&w_out_d, sizeof(float)*size*9));
    gpuErrchk(hipMalloc(&x_out_d, sizeof(float)*size*9));
    gpuErrchk(hipMalloc(&y_out_d, sizeof(float)*size*9));

    gpuErrchk(hipMalloc(&c_moments, sizeof(float)*size*7));
    gpuErrchk(hipMalloc(&mu, sizeof(float)*size*3));
    gpuErrchk(hipMalloc(&yf, sizeof(float)*size*3));

    gpuErrchk(hipMalloc(&m1, sizeof(float)*size*5));
    gpuErrchk(hipMalloc(&x1, sizeof(float)*size*3));
    gpuErrchk(hipMalloc(&w1, sizeof(float)*size*3));
    gpuErrchk(hipMalloc(&x2, sizeof(float)*size*3));
    gpuErrchk(hipMalloc(&w2, sizeof(float)*size*3));

    // Registers host memory as page-locked (required for asynch hipMemcpyAsync)
    gpuErrchk(hipHostRegister(moments, size*10*sizeof(float), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(w, size*9*sizeof(float), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(x, size*9*sizeof(float), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(y, size*9*sizeof(float), hipHostRegisterPortable));

    // Set up streams
    // Allocate 3 concurrent streams to each batch
    const int num_streams = batch_size*3;
    hipStream_t stream[num_streams];
    for (int i=0; i<num_streams; i++) {
        gpuErrchk(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
    }

    // Calculate optimal block and grid sizes
    int gridSize, blockSize;
    blockSize = 1024;
    gridSize = (size + blockSize - 1) / blockSize; 
    printf("[CHYQMOM9] Grid Size: %d Block Size: %d\n", gridSize, blockSize);
    // setup timer 
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    // hipProfilerStart();
    
    int size_per_batch = ceil(size / batch_size);
    printf("[CHYQMOM9] streams: %d size: %d, size_per_batch: %d\n",num_streams, size, size_per_batch);

    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    gpuErrchk(hipEventRecord(start));
    for (int i=0; i<num_streams; i+=3) {
        // beginning location in memory 
        int loc = (i/3) * size_per_batch;
        // transfer data from host to device 


        gpuErrchk(hipMemcpy2DAsync(&moments_d[loc], size*sizeof(float), 
                                    &moments[loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 10, 
                                    hipMemcpyHostToDevice, stream[i]));
        
        // Central moments
        chyqmom9_cmoments<<<gridSize, blockSize, 0, stream[i]>>>(&moments_d[loc], &c_moments[loc], size_per_batch, size);
        // setup first hyqmom3
        float_value_set<<<gridSize, blockSize, 0, stream[i+1]>>>(&m1[loc], 1, size_per_batch);
        float_value_set<<<gridSize, blockSize, 0, stream[i+2]>>>(&m1[size + loc], 0, size_per_batch);
        gpuErrchk(hipMemcpyAsync(&m1[2* size + loc], &c_moments[loc], size_per_batch*sizeof(float), 
                                    hipMemcpyDeviceToDevice, stream[i]));
        gpuErrchk(hipMemcpy2DAsync(&m1[3*size + loc], size*sizeof(float), 
                                    &c_moments[4*size + loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 2, 
                                    hipMemcpyDeviceToDevice, stream[i]));


        hyqmom3<<<gridSize, blockSize, 0, stream[i]>>>(&m1[loc], &x1[loc], &w1[loc], size_per_batch, size);
        // Compute mu and yf
        chyqmom9_mu_yf<<<gridSize, blockSize, 0, stream[i]>>>(&c_moments[loc], &x1[loc], &w1[loc], &yf[loc], &mu[loc], size_per_batch, size);
        // Set up second hyqmom3
        float_value_set<<<gridSize, blockSize, 0, stream[i+1]>>>(&m1[loc], 1, size_per_batch);
        float_value_set<<<gridSize, blockSize, 0, stream[i+2]>>>(&m1[size + loc], 0, size_per_batch);
        gpuErrchk(hipMemcpy2DAsync(&m1[2*size + loc], size*sizeof(float), 
                                    &mu[loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 3, 
                                    hipMemcpyDeviceToDevice, stream[i]));

        hyqmom3<<<gridSize, blockSize, 0, stream[i]>>>(&m1[loc], &x2[loc], &w2[loc], size_per_batch, size);
        // hipStreamSynchronize(stream[i]);
        // hipStreamSynchronize(stream[i+1]);
        // hipStreamSynchronize(stream[i+2]);

        // compute weight and copy data to host 
        chyqmom9_wout<<<gridSize, blockSize, 0, stream[i]>>>(&moments_d[loc], &w1[loc], &w2[loc], &w_out_d[loc], size_per_batch, size);
        gpuErrchk(hipMemcpy2DAsync(&w[loc], size*sizeof(float), 
                                    &w_out_d[loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 9, 
                                    hipMemcpyDeviceToHost, stream[i]));

        // compute x and copy data to host 
        chyqmom9_xout<<<gridSize, blockSize, 0, stream[i+1]>>>(&moments_d[loc], &x1[loc], &x_out_d[loc], size_per_batch, size);
        gpuErrchk(hipMemcpy2DAsync(&x[loc], size*sizeof(float), 
                                    &x_out_d[loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 9, 
                                    hipMemcpyDeviceToHost, stream[i+1]));
        // compute y and copy data to host 
        chyqmom9_yout<<<gridSize, blockSize, 0, stream[i+2]>>>(&moments_d[loc], &x2[loc], &yf[loc], &y_out_d[loc], size_per_batch, size);
        gpuErrchk(hipMemcpy2DAsync(&y[loc], size*sizeof(float), 
                                    &y_out_d[loc], size*sizeof(float),
                                    size_per_batch * sizeof(float), 9, 
                                    hipMemcpyDeviceToHost, stream[i+2]));
    }
    hipDeviceSynchronize();
    high_resolution_clock::time_point t2 = high_resolution_clock::now();


    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    float calc_duration;
    hipEventElapsedTime(&calc_duration, start, stop);
    printf("[CUDA] %f ms \n", calc_duration);
    printf("[CHRONO] %f ms \n", time_span.count()*1e3);
    // clean up
    hipFree(moments_d);
    hipFree(w_out_d);
    hipFree(x_out_d);
    hipFree(y_out_d);
    hipFree(c_moments);
    hipFree(mu);
    hipFree(yf);
    hipFree(x1);
    hipFree(x2);
    hipFree(w1);
    hipFree(w2);
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return calc_duration;
}