#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <cstdio>
#include <cassert>

//input and oytput are row majored

static __global__ void hyqmom2_naive(float mon[], float x[], float w[], int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < N) {
        // printf("hello from thread %d \n", idx);
        float C2 = ((mon[3*idx] * mon[3*idx+2]) - (mon[3*idx+1] * mon[3*idx+1])) 
                    / (mon[3*idx] * mon[3*idx]);
        w[2*idx] = mon[3*idx]/2;
        w[2*idx+1] = mon[3*idx]/2;
        x[2*idx] = (mon[3*idx+1]/mon[3*idx]) - sqrt(C2);
        x[2*idx+1] = (mon[3*idx+1]/mon[3*idx]) + sqrt(C2);
        idx += blockDim.x*gridDim.x;
    };
}

// input and output are column majored

static __global__ void hyqmom2_coalesced(float mon[], float x[], float w[], int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float mon_local[3];
    while (idx < N) {
        for (int i = 0; i < 3; i++) {
            mon_local[i] = mon[i * N + idx];
            // printf("thread [%d]: mon_local[%d] = %f \n", idx, i, mon_local[i]);
        }
        float C2 = ((mon_local[0] * mon_local[2]) - (mon_local[1] * mon_local[1])) 
            / (mon_local[0] * mon_local[0]);
        for (int i=0; i<2; i++) {
            w[i*N+idx] = mon_local[0]/2;
        }
        x[idx] = (mon_local[1]/mon_local[0]) - sqrt(C2);
        x[N + idx] = (mon_local[1]/mon_local[0]) + sqrt(C2);
        idx += blockDim.x*gridDim.x;
    }
}


float run_naive(const float moment[], const int size, float x_out[], float w_out[]) {
    float *x_out_cuda, *w_out_cuda, *moment_d;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMalloc(&x_out_cuda, sizeof(float)*size*2);
    hipMalloc(&moment_d, sizeof(float)*size*3);
    hipMalloc(&w_out_cuda, sizeof(float)*size*2);

    hipMemcpy(moment_d, moment, sizeof(float)*size*3, hipMemcpyHostToDevice);

    int gridSize = ceil(size/1024);
    hipEventRecord(start);
    hyqmom2_naive<<<gridSize, 1024>>>(moment_d, x_out_cuda, w_out_cuda, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(x_out, x_out_cuda, sizeof(float)*size*2, hipMemcpyDeviceToHost);
    hipMemcpy(w_out, w_out_cuda, sizeof(float)*size*2, hipMemcpyDeviceToHost);

    float calc_duration; 
    hipEventElapsedTime(&calc_duration, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(x_out_cuda);
    hipFree(w_out_cuda);
    hipFree(moment_d);

    return calc_duration;
}

float run_coal(const float moment[], const int size, float x_out[], float w_out[]) {
    float *x_out_cuda, *w_out_cuda, *moment_d;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&x_out_cuda, sizeof(float)*size*2);
    hipMalloc(&moment_d, sizeof(float)*size*3);
    hipMalloc(&w_out_cuda, sizeof(float)*size*2);

    hipMemcpy(moment_d, moment, sizeof(float)*size*3, hipMemcpyHostToDevice);

    int gridSize = ceil(size/1024);
    hipEventRecord(start);
    hyqmom2_coalesced<<<gridSize, 1024>>>(moment_d, x_out_cuda, w_out_cuda, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(x_out, x_out_cuda, sizeof(float)*size*2, hipMemcpyDeviceToHost);
    hipMemcpy(w_out, w_out_cuda, sizeof(float)*size*2, hipMemcpyDeviceToHost);

    float calc_duration; 
    hipEventElapsedTime(&calc_duration, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(x_out_cuda);
    hipFree(w_out_cuda);
    hipFree(moment_d);

    return calc_duration;
}
